#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"

__global__ void device_matmul(double *d_mat, double *d_kernel, double *d_res)
{
  //This kernel calculates convolution GPU.
  //Please modify this kernel!!
  int x = threadIdx.x;
  int y = blockIdx.x;
  int num = 1024;
  int width = num+2;
  double tmp_sum = 0.0f;
  for (int ky=-1; ky<2; ky++) 
      for (int kx=-1; kx<2; kx++)
        tmp_sum += d_kernel[ (ky+1)*3 + kx+1] * d_mat[(y+ky)*width + x + kx];
      
  d_res[y*width + x] = tmp_sum;
      
}

__host__ void launch_kernel(int num, double *gpu_mat, double *gpu_convkernel, double *gpu_matDst)
{

  //This function launches the gpu-kernel (a kind of function).
  //Please modify this function for convolutional calculation.
  //You need to allocate the device memory and so on in this function.


  ////////// initialization //////////
  int width = num+2;
  double **tmpmat_2 = (double**) malloc(sizeof(double*) * (num+2));
  for (int i=0; i<num+2; i++)  {
    tmpmat_2[i] = (double*)malloc(sizeof(double) * (num+2));
  }
  for (int i=0; i<num+2; i++)  {
    tmpmat_2[0][i] = 0.0f;
    tmpmat_2[num+1][i] = 0.0f;
  }
  
  for (int i=1; i<=num; i++)  {
    tmpmat_2[i][0] = 0.0f;
    for (int j=1; j<=num; j++) {
      tmpmat_2[i][j] = gpu_mat[(i-1)*num + (j-1)];
    }
    tmpmat_2[i][num+1] = 0.0f;
  }
  
  
  
  
  
  double *tmpmat = (double *)malloc(sizeof(double) * width*width);
  for (int i=1; i<=num; i++)  {
    tmpmat[i*width + 0] = 0.0f;
    for (int j=1; j<=num; j++) {
      tmpmat[i*width + j]  = gpu_mat[(i-1)*num + (j-1)];
    }
    tmpmat[i*width + num+1] = 0.0f;
  }
  for (int i=0; i<num+2; i++){
    //printf("%d | %d | MAX: %d\n", i, (num+1)*width + i, (num+2)*(num+2));
    tmpmat[0*width + i] = 0.0f;
    tmpmat[(num+1)*width + i] = 0.0f;
  }
  
  
  double *d_mat, *d_kernel, *d_res;
  hipMalloc((void **)&d_mat, sizeof(double) * (num+2)* (num+2));
  hipMalloc((void **)&d_kernel, sizeof(double) * 9);
  hipMalloc((void **)&d_res, sizeof(double) * num*num);
  
  hipMemcpy(d_mat, tmpmat, sizeof(double) * width*width, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, gpu_convkernel, sizeof(double) * 9, hipMemcpyHostToDevice);

  for( int x=0; x<num; x++)
  for( int y=0; y<num; y++){
    double tmp_sum = 0.0f;
    double tmp_sum_2 = 0.0f;
    for (int ky=0; ky<3; ky++) 
        for (int kx=0; kx<3; kx++){
            tmp_sum += gpu_convkernel[ ky*3 + kx] * tmpmat[(y+ky)*width + x+kx];
            //tmp_sum_2 += gpu_convkernel[ ky*3 + kx] * tmpmat_2[y + ky][x + kx];
        }
    //gpu_matDst[y*width + x] = tmp_sum;
    gpu_matDst[ y*num + x ] = tmp_sum;
  }

  //device_matmul<<<1024, 1024>>>(d_mat, d_kernel, d_res);
  //hipMemcpy(gpu_matDst, d_res, sizeof(double) * width*width, hipMemcpyDeviceToHost);

  
  
  ////////////////////////////////////
  /*
  for (int i=1; i<=num; i++) {
    for (int j=1; j<=num; j++) {
      double tmpsum = 0.0f;
      for (int ky=0; ky<3; ky++) 
      for (int kx=0; kx<3; kx++)
        tmpsum += gpu_convkernel[ ky*3 + kx] * tmpmat_2[i-1 + ky][j-1 + kx];
        
      gpu_matDst[ (i-1)*num + j-1 ] = tmpsum;
    }
  }
 */

  // ------free------ // 
  for (int i=0; i<num+2; i++)  {
    free(tmpmat_2[i]);
  }
  free(tmpmat);
  free(tmpmat_2);
  hipFree(d_mat);
  hipFree(d_kernel);
  hipFree(d_res);
  

}



