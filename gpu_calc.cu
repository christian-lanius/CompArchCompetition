#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"
#include <unistd.h>
#include <hip/hip_runtime_api.h>

#define NUM_ROWS 2
#define NUM_STREAMS 4
__global__ void device_matmul( int num, int stream_offset, double *gpu_in, double *gpu_kernel, double *gpu_out)
{
  //This kernel calculates convolution GPU.
  //Please modify this kernel!!

  int x;
  int y;
  x = threadIdx.x;
  y = NUM_ROWS*blockIdx.x+stream_offset*num/NUM_STREAMS;


  extern __shared__ double s[];

  #pragma unroll
  for(int offset=0; offset<(2+NUM_ROWS);offset++){
    s[offset*(num+2) + x] = gpu_in[(y + offset)*(num+2) + x];  
  }
  if(x >= num - 2){
    #pragma unroll
    for(int offset=0; offset<(2+NUM_ROWS);offset++){
      s[offset*(num+2) + x+2] = gpu_in[(y + offset)*(num+2) + x+2];  
    }
    
  }
  __syncthreads();
  
  for(int offset=0;offset<NUM_ROWS;offset++){
    double tmpsum = 0.0f;
    #pragma unroll
    for (int ky=0; ky<3; ky++){ 
      #pragma unroll
      for (int kx=0; kx<3; kx++){
        tmpsum += gpu_kernel[ ky*3 + kx] * s[(ky+offset)*(num+2) + (x + kx)];
      }
    }
    //printf("(%d|%d)\n", x,y+offset);
    gpu_out[ (y+offset)*num + x ] = tmpsum;
  }

}

__host__ void launch_kernel(int num, double *gpu_mat, double *gpu_convkernel, double *gpu_matDst)
{

  //This function launches the gpu-kernel (a kind of function).
  //Please modify this function for convolutional calculation.
  //You need to allocate the device memory and so on in this function.

  ////////// initialization //////////
  hipProfilerStart();
  double *gpu_in;
  double *gpu_out;
  double *gpu_kernel;
  //double *out_pinned;
  //hipHostMalloc((void **) &out_pinned, sizeof(double) * num * num);
  hipMalloc((void **) &gpu_in, sizeof(double) * (num+2) * (num+2));
  hipMemset(gpu_in, 0, sizeof(double) * (num+2)* (num+2));
  
  
  //Kernel initalization
  hipMalloc((void **) &gpu_kernel, sizeof(double) * 3*3);
  hipMemcpyAsync(gpu_kernel, gpu_convkernel, sizeof(double) * 3*3, hipMemcpyHostToDevice);
  //Input and Output Initalization
  
  hipMalloc((void **) &gpu_out, sizeof(double) * num * num);
  
  
  ////////////////////////////////////
  hipStream_t streams[NUM_STREAMS];
  for(int stream_idx=0;stream_idx<NUM_STREAMS;stream_idx++){
    hipStreamCreate(&streams[stream_idx]);
  }

  for (int i=1; i<=num; i++)  {
    hipMemcpyAsync(&gpu_in[i*(num+2)+1], &gpu_mat[(i-1)*num], sizeof(double)*(num), hipMemcpyHostToDevice, streams[i%NUM_STREAMS]);
  }
  
  for(int stream_idx=0;stream_idx<NUM_STREAMS;stream_idx++){
    int offset = stream_idx*num*num/NUM_STREAMS;
    device_matmul<<<num/NUM_ROWS/NUM_STREAMS,num, (2+NUM_ROWS)*(num+2)*sizeof(double), streams[stream_idx]>>>(num, stream_idx, gpu_in, gpu_kernel, gpu_out);
    hipMemcpyAsync(&gpu_matDst[offset], &gpu_out[offset], sizeof(double) * num * num/NUM_STREAMS, hipMemcpyDeviceToHost, streams[stream_idx]);
  }

  hipDeviceSynchronize();
  //memcpy(gpu_matDst, out_pinned, sizeof(double)*num*num);
  //gpu_matDst = out_pinned;
  
  
  
  // ------free------ // 
  return;
  hipFree(gpu_in);
  hipFree(gpu_kernel);
  hipFree(gpu_out);
  //free(tmpmat);
  //hipProfilerStop();
  

}





