#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"

__global__ void device_matmul(double *d_mat, double *d_kernel, double *d_res)
{
  //This kernel calculates convolution GPU.
  //Please modify this kernel!!
  int x = threadIdx.x;
  int y = blockIdx.x;
  int num = 1024;
  int width = num+2;
  double tmp_sum = 0.0f;
  for (int ky=-1; ky<2; ky++) 
      for (int kx=-1; kx<2; kx++)
        tmp_sum += d_kernel[ (ky+1)*3 + kx+1] * d_mat[(y+ky)*width + x + kx];
      
  d_res[y*width + x] = tmp_sum;
      
}

__host__ void launch_kernel(int num, double *gpu_mat, double *gpu_convkernel, double *gpu_matDst)
{

  //This function launches the gpu-kernel (a kind of function).
  //Please modify this function for convolutional calculation.
  //You need to allocate the device memory and so on in this function.


  ////////// initialization //////////
  
  //double **tmpmat = (double**) malloc(sizeof(double*) * (num+2));
  int width = num+2;
  double *tmpmat = (double *)malloc(sizeof(double) * width*width);
  //for (int i=0; i<num+2; i++)  {
  //  tmpmat[i] = (double*)malloc(sizeof(double) * (num+2));
  //}
  //for (int i=0; i<num+2; i++)  {
  //  tmpmat[0][i] = 0.0f;
  //  tmpmat[num+1][i] = 0.0f;
  //}
  for (int i=0; i<num+2; i++){
    //printf("%d | %d | MAX: %d\n", i, (num+1)*width + i, (num+2)*(num+2));
    tmpmat[0*width + i] = 0.0f;
    tmpmat[(num+1)*width + i] = 0.0f;
  }



  //for (int i=1; i<=num; i++)  {
  //  tmpmat[i][0] = 0.0f;
  //  for (int j=1; j<=num; j++) {
  //    tmpmat[i][j] = gpu_mat[(i-1)*num + (j-1)];
  //  }
  //  tmpmat[i][num+1] = 0.0f;
  //}
  
  
  for (int i=1; i<=num; i++)  {
    //tmpmat[i][0] = 0.0f;
    tmpmat[i*width + 0] = 0.0f;
    for (int j=1; j<=num; j++) {
      tmpmat[i*width + j]  = gpu_mat[(i-1)*num + (j-1)];
    }
    tmpmat[i*width + num+1] = 0.0f;
  }
  
  
  double *d_mat, *d_kernel, *d_res;
  hipMalloc((void **)&d_mat, sizeof(double) * (num+2)* (num+2));
  hipMalloc((void **)&d_kernel, sizeof(double) * 9);
  hipMalloc((void **)&d_res, sizeof(double) * num*num);
  
  hipMemcpy(d_mat, tmpmat, sizeof(double) * width*width, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, gpu_convkernel, sizeof(double) * 9, hipMemcpyHostToDevice);

  for( int x=0; x<num; x++)
  for( int y=0; y<num; y++){
    double tmp_sum = 0.0f;
    //printf("output: %d|%d\n", x,y);
    for (int ky=0; ky<3; ky++) 
        for (int kx=0; kx<3; kx++){
            //printf("x: %d | y: %d\n", x+ kx, (y+ky));
            tmp_sum += gpu_convkernel[ ky*3 + kx] * tmpmat[(y+ky)*width + x+kx];
        }
    //printf("%f\n",tmp_sum);
    //printf("-----------------\n");
    gpu_matDst[y*width + x] = tmp_sum;
  }
  
  
  
  //device_matmul<<<1024, 1024>>>(d_mat, d_kernel, d_res);
  //hipMemcpy(gpu_matDst, d_res, sizeof(double) * width*width, hipMemcpyDeviceToHost);

  ////////////////////////////////////
  /*
  for (int i=1; i<=num; i++) {
    for (int j=1; j<=num; j++) {
      double tmpsum = 0.0f;
      for (int ky=0; ky<3; ky++) 
      for (int kx=0; kx<3; kx++)
        tmpsum += gpu_convkernel[ ky*3 + kx] * tmpmat[i-1 + ky][j-1 + kx];
        
      gpu_matDst[ (i-1)*num + j-1 ] = tmpsum;
    }
  }
 */

  // ------free------ // 
  //for (int i=0; i<num+2; i++)  {
  //  free(tmpmat[i]);
  //}
  free(tmpmat);
  hipFree(d_mat);
  hipFree(d_kernel);
  hipFree(d_res);
  

}



