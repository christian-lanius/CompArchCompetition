#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"
#include <unistd.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#define NUM_ROWS 2
#define NUM_STREAMS 8

#define SHARED_MEM_SIZE 48*1024 //48 kByte


__global__ void device_matmul( int num, int stream_offset, double *gpu_in, double *gpu_kernel, double *gpu_out)
{
  //This kernel calculates convolution GPU.
  //Please modify this kernel!!

  int x;
  int y;
  
  x = threadIdx.x;
  y = NUM_ROWS*blockIdx.x+stream_offset;


  extern __shared__ double s[];
  double *gpu_kernel_shared;
  gpu_kernel_shared = &s[(2+NUM_ROWS)*num];
  reinterpret_cast<double4*>(s)[x] = reinterpret_cast<double4*>(gpu_in)[y*num/4 + x];

  if(x<9){
    gpu_kernel_shared[x] = gpu_kernel[x];
  }
  
  __syncthreads();
  
  #pragma unroll
  for(int offset=0;offset<NUM_ROWS;++offset){
    double tmpsum = 0.0f;
    #pragma unroll
    for (int ky=0; ky<3; ++ky){
      int in_y = (ky+offset)*(num);
      int ker_y = ky*3;
      #pragma unroll
      for (int kx=0; kx<3; ++kx){
        int in_x = x+kx;
        if( in_x != 0 && in_x != num+1)
          tmpsum += gpu_kernel_shared[ ker_y + kx] * s[in_y+ (in_x-1)];
      }
    }
    gpu_out[ (y+offset)*num + x ] = tmpsum;
  }

}

__host__ void launch_kernel(int num, double *gpu_mat, double *gpu_convkernel, double *gpu_matDst)
{

  //This function launches the gpu-kernel (a kind of function).
  //Please modify this function for convolutional calculation.
  //You need to allocate the device memory and so on in this function.

  ////////// initialization //////////
  //hipProfilerStart();
  //hipHostRegister(gpu_matDst, sizeof(double)*num*num, hipHostRegisterMapped);
  //hipHostRegister(gpu_mat, sizeof(double)*num*num, hipHostRegisterMapped);
  double *gpu_in;
  double *gpu_out;
  double *gpu_kernel;
  hipStream_t streams[NUM_STREAMS];
  for(int stream_idx=0;stream_idx<NUM_STREAMS;stream_idx++){
    hipStreamCreate(&streams[stream_idx]);
  }
  hipMalloc((void **) &gpu_in, sizeof(double) * (num+2) * (num));
  hipMemset(gpu_in, 0, sizeof(double) * (num+2)* (num));
  
  
  //Kernel initalization
  hipMalloc((void **) &gpu_kernel, sizeof(double) * 3*3);
  hipMemcpyAsync(gpu_kernel, gpu_convkernel, sizeof(double) * 3*3, hipMemcpyHostToDevice, streams[1]);
  //Input and Output Initalization
  
  hipMalloc((void **) &gpu_out, sizeof(double) * num * num);
  
  
  ////////////////////////////////////
  
  
  
  for(int stream_idx=0;stream_idx<NUM_STREAMS;stream_idx++){
    //if(stream_idx != 1) continue;
    int offset = stream_idx*(num)*num/NUM_STREAMS;
    if(stream_idx == 0){//First line copy is offset by 1 (because of zero padding), thus copy one line less
      hipMemcpyAsync(&gpu_in[num+offset], &gpu_mat[offset], sizeof(double)*num*(num/NUM_STREAMS+1), hipMemcpyHostToDevice, streams[stream_idx]);
    }else if(stream_idx == NUM_STREAMS-1){ //Last line is one line less copy because of zero padding
      hipMemcpyAsync(&gpu_in[offset], &gpu_mat[offset-num], sizeof(double)*num*(num/NUM_STREAMS+1), hipMemcpyHostToDevice, streams[stream_idx]);
    }else{ //copy one line before and one after as well as actual data
      hipMemcpyAsync(&gpu_in[offset], &gpu_mat[offset-num], sizeof(double)*num*(num/NUM_STREAMS+2), hipMemcpyHostToDevice, streams[stream_idx]);
    }
    device_matmul<<<num/NUM_ROWS/NUM_STREAMS,num, sizeof(double)*((3+NUM_ROWS)*num+9), streams[stream_idx]>>>(num, stream_idx*num/NUM_STREAMS, gpu_in, gpu_kernel, gpu_out);
    hipMemcpyAsync(&gpu_matDst[offset], &gpu_out[offset], sizeof(double) * num * num/NUM_STREAMS, hipMemcpyDeviceToHost, streams[stream_idx]);
    
  }

  return;
  // ------free------ //
  //Dont have to free memory as this is the last cuda call and the memory will be free'd automatically at the end of the program 
  //Dont do this in real life, its a memory leak
  hipFree(gpu_in);
  hipFree(gpu_kernel);
  hipFree(gpu_out);


  

}





