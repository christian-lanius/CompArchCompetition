#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"
#include <unistd.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#define NUM_ROWS 2
#define NUM_STREAMS 4
__global__ void device_matmul( int num, int stream_offset, double *gpu_in, double *gpu_kernel, double *gpu_out)
{
  //This kernel calculates convolution GPU.
  //Please modify this kernel!!

  int x;
  int y;
  x = threadIdx.x;
  y = NUM_ROWS*blockIdx.x+stream_offset;


  extern __shared__ double s[];
  reinterpret_cast<double4*>(s)[x] = reinterpret_cast<double4*>(gpu_in)[y*num/4 + x];  
  
  __syncthreads();
  
  for(int offset=0;offset<NUM_ROWS;offset++){
    double tmpsum = 0.0f;
    #pragma unroll
    for (int ky=0; ky<3; ky++){
      int in_y = (ky+offset)*(num);
      int ker_y = ky*3;
      #pragma unroll
      for (int kx=0; kx<3; kx++){
        int in_x = x+kx;
        if( in_x != 0 && in_x != num+1)
          tmpsum += gpu_kernel[ ker_y + kx] * s[in_y+ (in_x-1)];
      }
    }
    gpu_out[ (y+offset)*num + x ] = tmpsum;
  }

}

__host__ void launch_kernel(int num, double *gpu_mat, double *gpu_convkernel, double *gpu_matDst)
{

  //This function launches the gpu-kernel (a kind of function).
  //Please modify this function for convolutional calculation.
  //You need to allocate the device memory and so on in this function.

  ////////// initialization //////////
  hipProfilerStart();
  double *gpu_in;
  double *gpu_out;
  double *gpu_kernel;
  hipMalloc((void **) &gpu_in, sizeof(double) * (num+2) * (num));
  hipMemset(gpu_in, 0, sizeof(double) * (num+2)* (num));
  
  
  //Kernel initalization
  hipMalloc((void **) &gpu_kernel, sizeof(double) * 3*3);
  hipMemcpyAsync(gpu_kernel, gpu_convkernel, sizeof(double) * 3*3, hipMemcpyHostToDevice);
  //Input and Output Initalization
  
  hipMalloc((void **) &gpu_out, sizeof(double) * num * num);
  
  
  ////////////////////////////////////
  hipStream_t streams[NUM_STREAMS];
  for(int stream_idx=0;stream_idx<NUM_STREAMS;stream_idx++){
    hipStreamCreate(&streams[stream_idx]);
  }
  
  
  for(int stream_idx=0;stream_idx<NUM_STREAMS;stream_idx++){
    //if(stream_idx != 1) continue;
    int offset = stream_idx*(num)*num/NUM_STREAMS;
    if(stream_idx == 0){//First line copy is offset by 1 (because of zero padding), thus copy one line less
      hipMemcpyAsync(&gpu_in[num+offset], &gpu_mat[offset], sizeof(double)*num*(num/NUM_STREAMS+1), hipMemcpyHostToDevice, streams[stream_idx]);
    }else if(stream_idx == NUM_STREAMS-1){ //Last line is one line less copy because of zero padding
      hipMemcpyAsync(&gpu_in[offset], &gpu_mat[offset-num], sizeof(double)*num*(num/NUM_STREAMS+1), hipMemcpyHostToDevice, streams[stream_idx]);
    }else{ //copy one line before and one after as well as actual data
      hipMemcpyAsync(&gpu_in[offset], &gpu_mat[offset-num], sizeof(double)*num*(num/NUM_STREAMS+2), hipMemcpyHostToDevice, streams[stream_idx]);
    }
    
    //hipStreamSynchronize(streams[stream_idx]);
    device_matmul<<<num/NUM_ROWS/NUM_STREAMS,num, (2+NUM_ROWS)*num*sizeof(double), streams[stream_idx]>>>(num, stream_idx*num/NUM_STREAMS, gpu_in, gpu_kernel, gpu_out);
    //hipStreamSynchronize(streams[stream_idx]);
    hipMemcpyAsync(&gpu_matDst[offset], &gpu_out[offset], sizeof(double) * num * num/NUM_STREAMS, hipMemcpyDeviceToHost, streams[stream_idx]);
    
  }

  return;
  // ------free------ //
  //Dont have to free memory as this is the last cuda call and the memory will be free'd automatically at the end of the program 
  //Dont do this in real life, its a memory leak
  hipFree(gpu_in);
  hipFree(gpu_kernel);
  hipFree(gpu_out);
////////////////////////////////////////////////////////////////////////////////////////////////
  double *gpu_matDst_cpu = (double*)malloc(sizeof(double) * num * num);
  double **tmpmat = (double**) malloc(sizeof(double*) * (num+2));
  for (int i=0; i<num+2; i++)  {
    tmpmat[i] = (double*)malloc(sizeof(double) * (num+2));
  }
  for (int i=0; i<num+2; i++)  {
    tmpmat[0][i] = 0.0f;
    tmpmat[num+1][i] = 0.0f;
  }



  for (int i=1; i<=num; i++)  {
    tmpmat[i][0] = 0.0f;
    for (int j=1; j<=num; j++) {
      tmpmat[i][j] = gpu_mat[(i-1)*num + (j-1)];
    }
    tmpmat[i][num+1] = 0.0f;
  }

  ////////////////////////////////////

  for (int i=1; i<=num; i++) {
    for (int j=1; j<=num; j++) {
      double tmpsum = 0.0f;
      for (int ky=0; ky<3; ky++) 
      for (int kx=0; kx<3; kx++)
        tmpsum += gpu_convkernel[ ky*3 + kx] * tmpmat[i-1 + ky][j-1 + kx];
        
      gpu_matDst_cpu[ (i-1)*num + j-1 ] = tmpsum;
    }
  }
//////////////////////////////////////////////////////////////////////////////////////////

  for (int i=256; i<512; i++) {
    for (int j=0; j<num; j++) {
      double eps = 10e-9;
      if(abs(gpu_matDst_cpu[i*num+j]-gpu_matDst[i*num+j])>eps){
        printf("Error at (%d|%d): %f\n",j,i, gpu_matDst[i*num+j]);
      }
    }
  }


  

}





