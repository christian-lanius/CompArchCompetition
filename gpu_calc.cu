#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"

__global__ void device_matmul( int num, double *gpu_int, double *gpu_kernel, double *gpu_out)
{
  //This kernel calculates convolution GPU.
  //Please modify this kernel!!
}

__host__ void launch_kernel(int num, double *gpu_mat, double *gpu_convkernel, double *gpu_matDst)
{

  //This function launches the gpu-kernel (a kind of function).
  //Please modify this function for convolutional calculation.
  //You need to allocate the device memory and so on in this function.


  ////////// initialization //////////
  
  double **tmpmat = (double**) malloc(sizeof(double*) * (num+2));
  double *tmpArray = (double *) malloc(sizeof(double) * (num+2) * (num+2));
  for (int i=0; i<num+2; i++)  {
    tmpmat[i] = &tmpArray[i*(num+2)];
  }
  
  memset(tmpmat[0], 0, sizeof(double) * (num+2));
  memset(tmpmat[num+1], 0, sizeof(double) * (num+2));
  for (int i=1; i<=num; i++)  {
    tmpmat[i][0] = 0.0f;
    memcpy( &(tmpmat[i][1]), &gpu_mat[(i-1)*num], sizeof(double)*num);
    tmpmat[i][num+1] = 0.0f;
  }
  
  ////////////////////////////////////

  for (int i=1; i<=num; i++) {
    for (int j=1; j<=num; j++) {
      double tmpsum = 0.0f;
      for (int ky=0; ky<3; ky++) 
      for (int kx=0; kx<3; kx++)
        tmpsum += gpu_convkernel[ ky*3 + kx] * tmpmat[i-1 + ky][j-1 + kx];
        
      gpu_matDst[ (i-1)*num + j-1 ] = tmpsum;
    }
  }
  
  
  // ------free------ // 
  /*
  for (int i=0; i<num+2; i++)  {
    free(tmpmat[i]);
  }*/
  free(tmpArray);
  free(tmpmat);

  

}



