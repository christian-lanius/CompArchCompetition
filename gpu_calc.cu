#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"
#include <unistd.h>


__global__ void device_matmul( int num, double *gpu_in, double *gpu_kernel, double *gpu_out)
{
  //This kernel calculates convolution GPU.
  //Please modify this kernel!!

  int x;
  int y;
  x = threadIdx.x;
  y = blockIdx.x;

  extern __shared__ double s[];
  s[0*(num+2) + x] = gpu_in[(y + 0)*(num+2) + x];
  s[1*(num+2) + x] = gpu_in[(y + 1)*(num+2) + x];
  s[2*(num+2) + x] = gpu_in[(y + 2)*(num+2) + x];
  
  if(x >= num - 2){
    s[0*(num+2) + x+2] = gpu_in[(y + 0)*(num+2) + x+2];
    s[1*(num+2) + x+2] = gpu_in[(y + 1)*(num+2) + x+2];
    s[2*(num+2) + x+2] = gpu_in[(y + 2)*(num+2) + x+2];
    
  }
  __syncthreads();

  double tmpsum = 0.0f;
  #pragma unroll
  for (int ky=0; ky<3; ky++){ 
    #pragma unroll
    for (int kx=0; kx<3; kx++){
      tmpsum += gpu_kernel[ ky*3 + kx] * s[ky*(num+2) + (x + kx)];
    }
  }
  //printf("(%d|%d)\n", x,y);
  gpu_out[ y*num + x ] = tmpsum;

}

__host__ void launch_kernel(int num, double *gpu_mat, double *gpu_convkernel, double *gpu_matDst)
{

  //This function launches the gpu-kernel (a kind of function).
  //Please modify this function for convolutional calculation.
  //You need to allocate the device memory and so on in this function.


  ////////// initialization //////////

  double *gpu_in;
  double *gpu_out;
  double *gpu_kernel;
  
  //Kernel initalization
  hipMalloc((void **) &gpu_kernel, sizeof(double) * 3*3);
  hipMemcpyAsync(gpu_kernel, gpu_convkernel, sizeof(double) * 3*3, hipMemcpyHostToDevice);
  //Input and Output Initalization
  hipMalloc((void **) &gpu_in, sizeof(double) * (num+2) * (num+2));
  hipMalloc((void **) &gpu_out, sizeof(double) * num * num);
  hipMemset(gpu_in, 0, sizeof(double) * (num+2)* (num+2));
  for (int i=1; i<=num; i++)  {
    hipMemcpyAsync(&gpu_in[i*(num+2)+1], &gpu_mat[(i-1)*num], sizeof(double)*(num), hipMemcpyHostToDevice);
  }
  
  ////////////////////////////////////
  
  device_matmul<<<num,num, 3*(num+2)*sizeof(double)>>>(num, gpu_in, gpu_kernel, gpu_out);
  hipMemcpy(gpu_matDst, gpu_out, sizeof(double) * num * num, hipMemcpyDeviceToHost);
  
  
  // ------free------ // 
  //free(tmpArray);
  //free(tmpmat);
  
  

}





